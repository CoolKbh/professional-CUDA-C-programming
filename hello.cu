
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void helloFromGPU(void){
    printf("Hello world from GPU thread %d!\n", threadIdx.x);
}

int main(void){
    //hello from CPU
    printf("Hello world from CPU!\n");

    //hello from GPU
    helloFromGPU<<<2, 10>>>();
    hipDeviceSynchronize();
    //cudaDeviceReset();
    return 0;
}
